#include <cstdio>
#include <cstdlib>
#include <string>
#include <ctime>
#include <sys/time.h>
#include <cmath>
#include "hip/hip_fp16.h"
#include <mma.h>
#include <hip/hip_runtime.h>

#define cdiv(x, y) (((x) + (y) - 1) / (y))

__global__ void base_kernel(half* d_A, half* d_B, half* d_C, int M, int N, int K) {
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.0;
    for (int k = 0; k < K; k++) {
        sum += __half2float(d_A[m * K + k]) * __half2float(d_B[k * N + n]);
    }
    d_C[m * N + n] = __float2half(sum);
}

void baseline(int M, int N, int K, half* h_A, half* h_B, half* h_C) {
    half* d_A;
    half* d_B;
    half* d_C;
    struct timeval tv;
    double start, end;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(half), hipMemcpyHostToDevice);

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    base_kernel<<<dim3(cdiv(M, 16), cdiv(N, 16)), dim3(16, 16)>>>(d_A, d_B, d_C, M, N, K);

    hipDeviceSynchronize();
    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("baseline time: %lf\n", end - start);

    hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}