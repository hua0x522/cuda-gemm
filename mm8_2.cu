#include "hip/hip_runtime.h"
/*
    double-warp multi-tile
*/
#include <cstdio>
#include <cstdlib>
#include <string>
#include <ctime>
#include <sys/time.h>
#include <cmath>
#include "hip/hip_fp16.h"
#include <mma.h>
#include <hip/hip_runtime.h>
#include <ptx.h>

#define cdiv(x, y) (((x) + (y) - 1) / (y))

/*
    block size: [64, 64]
    warp  size: [64, 32]
    warp  num : [ 1,  2]
    k : 64
*/

__device__ void load_shm_A(half* shm_A, half* A, int M, int K, int ko) {
    // layout: [64, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 8; i++) {
        int row = i * 8 + tid / 8;
        int col = tid % 8 * 8;
        *(float4*)&shm_A[row * 64 + col] = *(float4*)&A[(blockIdx.x * 64 + row) * K + ko * 64 + col];
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [64, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 8; i++) {
        int row = i * 8 + tid / 8;
        int col = tid % 8 * 8;
        *(float4*)&shm_B[row * 64 + col] = *(float4*)&B[(ko * 64 + row) * N + blockIdx.y * 64 + col];
    }
    __syncthreads();
}

__device__ void store_shm_C(float* shm_C, half* C, int M, int N) {
    // layout: [64, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 64; i++) {
        int row = i;
        int col = tid;
        C[(blockIdx.x * 64 + row) * N + blockIdx.y * 64 + col] = __float2half(shm_C[row * 64 + col]);
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int mi, int ki) {
    int lane_id = threadIdx.x;
    uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + (mi * 16 + lane_id % 16) * 64 + ki * 16 + lane_id / 16 * 8);
    LDMATRIX_X4(reg_A[0], reg_A[1], reg_A[2], reg_A[3], shm_A_lane_addr);
    __syncthreads();

}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    for (int ni = 0; ni < 4; ni++) {
        uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + (ki * 16 + lane_id % 16) * 64 + threadIdx.y * 32 + ni * 8);
        LDMATRIX_X2_T(reg_B[ni * 2], reg_B[ni * 2 + 1], shm_B_lane_addr);
    }
}

__device__ void store_reg_C(uint32_t* reg_C, float* shm_C, int mi) {
    int lane_id = threadIdx.x;

    for (int ni = 0; ni < 4; ni++) {
        int idx = (mi * 16 + lane_id / 4) * 64 + threadIdx.y * 32 + ni * 8 + (lane_id % 4) * 2;
        shm_C[idx] += *(float*)(&reg_C[ni * 4]);
        shm_C[idx + 1] += *(float*)(&reg_C[ni * 4 + 1]);
        shm_C[idx + 8 * 64] += *(float*)(&reg_C[ni * 4 + 2]);
        shm_C[idx + 8 * 64 + 1] += *(float*)(&reg_C[ni * 4 + 3]);
    }
}

__device__ void clear_shm_C(float* shm_C) {
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 64; i++) {
        shm_C[i * 64 + tid] = 0;
    }
}

__device__ void clear_reg_C(uint32_t* reg_C) {
    for (int i = 0; i < 16; i++) {
        reg_C[i] = 0;
    }
}

__global__ void matmul_kernel(int M, int N, int K, half* d_A, half* d_B, half* d_C) {
    __shared__ half shm_A[64 * 16];
    __shared__ half shm_B[16 * 64];
    __shared__ float shm_C[64 * 64];

    uint32_t reg_A[4];
    uint32_t reg_B[4 * 2];
    uint32_t reg_C[4 * 4];
    clear_shm_C(shm_C);

    for (int k = 0; k < K / 64; k++) {
        load_shm_A(shm_A, d_A, M, K, k);
        load_shm_B(shm_B, d_B, K, N, k);
        __syncthreads();

        for (int m = 0; m < 4; m++) {
            for (int ki = 0; ki < 4; ki++) {
                load_reg_A(reg_A, shm_A, m, ki);
                load_reg_B(reg_B, shm_B, ki);
                __syncthreads();

                for (int n = 0; n < 4; n++) {
                    HMMA16816(reg_C[n * 4], reg_C[n * 4 + 1], reg_C[n * 4 + 2], reg_C[n * 4 + 3],
                              reg_A[0], reg_A[1], reg_A[2], reg_A[3],
                              reg_B[n * 2], reg_B[n * 2 + 1],
                              reg_C[n * 4], reg_C[n * 4 + 1], reg_C[n * 4 + 2], reg_C[n * 4 + 3]);
                }
            }
            __syncthreads();
            store_reg_C(reg_C, shm_C, m);
        }
    }
    store_shm_C(shm_C, d_C, M, N);
}

void matmul(int M, int N, int K, half* h_A, half* h_B, half* h_C) {
    half* d_A;
    half* d_B;
    half* d_C;
    struct timeval tv;
    double start, end;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(half), hipMemcpyHostToDevice);

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    matmul_kernel<<<dim3(cdiv(M, 64), cdiv(N, 64)), dim3(32, 2)>>>(M, N, K, d_A, d_B, d_C);

    hipDeviceSynchronize();
    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("matmul time: %lf\n", end - start);

    hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}