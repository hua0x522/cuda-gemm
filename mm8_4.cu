#include "hip/hip_runtime.h"
/*
    double-warp multi-tile
*/
#include <cstdio>
#include <cstdlib>
#include <string>
#include <ctime>
#include <sys/time.h>
#include <cmath>
#include "hip/hip_fp16.h"
#include <mma.h>
#include <hip/hip_runtime.h>
#include <ptx.h>

#define cdiv(x, y) (((x) + (y) - 1) / (y))

/*
    block size: [64, 64]
    warp  size: [64, 16]
    warp  num : [ 1,  4]
    k : 32
*/

__device__ void load_shm_A(half* shm_A, half* A, int M, int K, int ko) {
    // layout: [64, 40]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 32 + tid / 4;
        int col = tid % 4 * 8;
        *(float4*)&shm_A[row * 40 + col] = *(float4*)&A[(blockIdx.x * 64 + row) * K + ko * 32 + col];
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [32, 72]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        *(float4*)&shm_B[row * 72 + col] = *(float4*)&B[(ko * 32 + row) * N + blockIdx.y * 64 + col];
    }
    __syncthreads();
}

__device__ void store_shm_C(float* shm_C, half* C, int M, int N) {
    // layout: [64, 72]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 32; i++) {
        int row = i * 2 + tid / 64;
        int col = tid % 64;
        C[(blockIdx.x * 64 + row) * N + blockIdx.y * 64 + col] = __float2half(shm_C[row * 72 + col]);
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int mi) {
    for (int ki = 0; ki < 2; ki++) {
        int lane_id = threadIdx.x;
        int row = mi * 16 + lane_id % 16;
        int col = ki * 16 + lane_id / 16 * 8;
        uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + row * 40 + col);
        LDMATRIX_X4(reg_A[ki * 4], reg_A[ki * 4 + 1], reg_A[ki * 4 + 2], reg_A[ki * 4 + 3], shm_A_lane_addr);
    }
    __syncthreads();
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    for (int ni = 0; ni < 2; ni++) {
        int row = ki * 16 + lane_id % 16;
        int col = threadIdx.y * 16 + ni * 8;
        uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 72 + col);
        LDMATRIX_X2_T(reg_B[ki * 4 + ni * 2], reg_B[ki * 4 + ni * 2 + 1], shm_B_lane_addr);
    }
    __syncthreads();
}

__device__ void store_reg_C(uint32_t* reg_C, float* shm_C, int mi) {
    int lane_id = threadIdx.x;

    for (int ni = 0; ni < 2; ni++) {
        int row = mi * 16 + lane_id / 4;
        int col = threadIdx.y * 16 + ni * 8 + (lane_id % 4) * 2;
        shm_C[row * 72 + col] += *(float*)(&reg_C[ni * 4]);
        shm_C[row * 72 + col + 1] += *(float*)(&reg_C[ni * 4 + 1]);
        shm_C[(row + 8) * 72 + col] += *(float*)(&reg_C[ni * 4 + 2]);
        shm_C[(row + 8) * 72 + col + 1] += *(float*)(&reg_C[ni * 4 + 3]);
    }
    __syncthreads();
}

__device__ void clear_shm_C(float* shm_C) {
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 36; i++) {
        shm_C[i * 128 + tid] = 0;
    }
    __syncthreads();
}

__device__ void clear_reg_C(uint32_t* reg_C) {
    for (int i = 0; i < 8; i++) {
        reg_C[i] = 0;
    }
    __syncthreads();
}

__global__ void matmul_kernel(int M, int N, int K, half* d_A, half* d_B, half* d_C) {
    __shared__ half shm_A[64 * 40];
    __shared__ half shm_B[32 * 72];
    __shared__ float shm_C[64 * 72];

    uint32_t reg_A[2 * 4];
    uint32_t reg_B[2 * 2 * 2];
    uint32_t reg_C[2 * 4];
    clear_shm_C(shm_C);

    for (int k = 0; k < K / 32; k++) {
        load_shm_A(shm_A, d_A, M, K, k);
        load_shm_B(shm_B, d_B, K, N, k);

        for (int ki = 0; ki < 2; ki++) {
            load_reg_B(reg_B, shm_B, ki);
        }

        for (int m = 0; m < 4; m++) {
            clear_reg_C(reg_C);
            load_reg_A(reg_A, shm_A, m);
            for (int ki = 0; ki < 2; ki++) {
                for (int n = 0; n < 2; n++) {
                    HMMA16816(reg_C[n * 4], reg_C[n * 4 + 1], reg_C[n * 4 + 2], reg_C[n * 4 + 3],
                              reg_A[ki * 4], reg_A[ki * 4 + 1], reg_A[ki * 4 + 2], reg_A[ki * 4 + 3],
                              reg_B[ki * 4 + n * 2], reg_B[ki * 4 + n * 2 + 1],
                              reg_C[n * 4], reg_C[n * 4 + 1], reg_C[n * 4 + 2], reg_C[n * 4 + 3]);
                }
            }
            store_reg_C(reg_C, shm_C, m);
        }
    }
    store_shm_C(shm_C, d_C, M, N);
}

void matmul(int M, int N, int K, half* h_A, half* h_B, half* h_C) {
    half* d_A;
    half* d_B;
    half* d_C;
    struct timeval tv;
    double start, end;

    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(half));
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(half), hipMemcpyHostToDevice);

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    matmul_kernel<<<dim3(cdiv(M, 64), cdiv(N, 64)), dim3(32, 4)>>>(M, N, K, d_A, d_B, d_C);

    hipDeviceSynchronize();
    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("matmul time: %lf\n", end - start);

    hipMemcpy(h_C, d_C, M * N * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}